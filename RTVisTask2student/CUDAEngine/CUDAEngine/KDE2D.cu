#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

__device__
float CudaIsotropicGaussKernel2D(float x, float y, float center_x, float center_y, float eps)
{
	register float tmpx = x - center_x;
	register float tmpy = y - center_y;
	register float eps_p_2 = eps*eps;

	return ::expf(-0.5f * tmpx * tmpx / eps_p_2) * ::expf(-0.5f * tmpy * tmpy / eps_p_2);
}

__global__
void CudaKDE2D(const float* xData, const float* yData, size_t dataCount, float epsilon, float minX, float maxX, float minY, float maxY, float* kdeImage, size_t numBins)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id >= static_cast<int>(numBins * numBins))
		return;

	int i = id / numBins;
	int j = id % numBins;

	float rangeX = maxX - minX;
	float rangeY = maxY - minY;
	float y = float(i) / (numBins - 1)*rangeY + minY;
	float x = float(j) / (numBins - 1)*rangeX + minX;

	float sum = 0.0f;
	for (int dataIndex = 0; dataIndex < dataCount; dataIndex++)
	{
		sum += CudaIsotropicGaussKernel2D(x, y, xData[dataIndex], yData[dataIndex], epsilon);
	}
	kdeImage[id] = sum;
}

float CallKDE2D(const float* xData, const float* yData, size_t dataCount, float epsilon, float minX, float maxX, float minY, float maxY, float* kdeImage, size_t numBins)
{
	size_t dataByteSize = dataCount * sizeof(float);
	size_t imageByteSize = numBins * numBins * sizeof(float);

	// Initialize GPU memory for the x-data:
	float* cudaXData;
	hipMalloc(&cudaXData, dataByteSize);
	hipMemcpy(cudaXData, xData, dataByteSize, hipMemcpyHostToDevice);

	// Initialize GPU memory for the y-data:
	float* cudaYData;
	hipMalloc(&cudaYData, dataByteSize);
	hipMemcpy(cudaYData, yData, dataByteSize, hipMemcpyHostToDevice);

	// Initialize GPU memory for KDE image with values 0:
	float* cudaKDEImage;
	hipMalloc(&cudaKDEImage, imageByteSize);
	hipMemset(cudaKDEImage, 0, imageByteSize);

	// Call kde 2d:
	int threadsPerBlock = 256;
	int blocksPerGrid = (numBins*numBins + threadsPerBlock - 1) / threadsPerBlock;
	CudaKDE2D << <blocksPerGrid, threadsPerBlock >> > (cudaXData, cudaYData, dataCount, epsilon, minX, maxX, minY, maxY, cudaKDEImage, numBins);

	// Copy memory of the KDE image from GPU to CPU:
	hipMemcpy(kdeImage, cudaKDEImage, imageByteSize, hipMemcpyDeviceToHost);

	// Free memory on GPU:
	hipFree(cudaKDEImage);
	hipFree(cudaYData);
	hipFree(cudaXData);

	return *std::max_element(kdeImage, kdeImage + (numBins*numBins));
}
