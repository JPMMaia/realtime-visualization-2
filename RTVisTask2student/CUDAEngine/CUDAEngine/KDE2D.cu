#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>

constexpr auto c_blockSize = 256;

__device__
float CudaIsotropicGaussKernel2D(float x, float y, float center_x, float center_y, float eps)
{
	register float tmpx = x - center_x;
	register float tmpy = y - center_y;
	register float eps_p_2 = eps*eps;

	return ::expf(-0.5f * tmpx * tmpx / eps_p_2) * ::expf(-0.5f * tmpy * tmpy / eps_p_2);
}

__global__
void CudaKDE2D(const float* xData, const float* yData, size_t dataCount, float epsilon, float minX, float maxX, float minY, float maxY, float* kdeImage, size_t numBins)
{
	int imageSize = static_cast<int>(numBins * numBins);
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id >= imageSize)
		return;

	int i = id / numBins;
	int j = id % numBins;

	float rangeX = maxX - minX;
	float rangeY = maxY - minY;
	float y = float(i) / (numBins - 1)*rangeY + minY;
	float x = float(j) / (numBins - 1)*rangeX + minX;

	float sum = 0.0f;

	// Version 1:
	/*{
		for (int dataIndex = 0; dataIndex < dataCount; dataIndex++)
		{
			sum += CudaIsotropicGaussKernel2D(x, y, xData[dataIndex], yData[dataIndex], epsilon);
		}
	}*/

	// Version 2:
	{
		__shared__ float xDataShared[c_blockSize];
		__shared__ float yDataShared[c_blockSize];

		for (int blockStart = 0; blockStart < dataCount; blockStart += c_blockSize)
		{
			for (int index = 0; index < c_blockSize && (blockStart + index) < dataCount; ++index)
			{
				xDataShared[index] = xData[blockStart + index];
				yDataShared[index] = yData[blockStart + index];
			}

			__syncthreads();

			for (int index = 0; index < c_blockSize && (blockStart + index) < dataCount; ++index)
			{
				sum += CudaIsotropicGaussKernel2D(x, y, xDataShared[index], yDataShared[index], epsilon);
			}

			__syncthreads();
		}
	}

	kdeImage[id] = sum;
}

float CallKDE2D(const float* xData, const float* yData, size_t dataCount, float epsilon, float minX, float maxX, float minY, float maxY, float* kdeImage, size_t numBins)
{
	size_t dataByteSize = dataCount * sizeof(float);
	size_t imageByteSize = numBins * numBins * sizeof(float);

	// Initialize GPU memory for the x-data:
	float* cudaXData;
	hipMalloc(&cudaXData, dataByteSize);
	hipMemcpy(cudaXData, xData, dataByteSize, hipMemcpyHostToDevice);

	// Initialize GPU memory for the y-data:
	float* cudaYData;
	hipMalloc(&cudaYData, dataByteSize);
	hipMemcpy(cudaYData, yData, dataByteSize, hipMemcpyHostToDevice);

	// Initialize GPU memory for KDE image with values 0:
	float* cudaKDEImage;
	hipMalloc(&cudaKDEImage, imageByteSize);
	hipMemset(cudaKDEImage, 0, imageByteSize);

	// Call kde 2d:
	int blockDimension = c_blockSize;
	int gridDimension = static_cast<int>((numBins*numBins + blockDimension - 1) / blockDimension);
	CudaKDE2D << <gridDimension, blockDimension >> > (cudaXData, cudaYData, dataCount, epsilon, minX, maxX, minY, maxY, cudaKDEImage, numBins);

	// Copy memory of the KDE image from GPU to CPU:
	hipMemcpy(kdeImage, cudaKDEImage, imageByteSize, hipMemcpyDeviceToHost);

	// Free memory on GPU:
	hipFree(cudaKDEImage);
	hipFree(cudaYData);
	hipFree(cudaXData);

	return *std::max_element(kdeImage, kdeImage + (numBins*numBins));
}
